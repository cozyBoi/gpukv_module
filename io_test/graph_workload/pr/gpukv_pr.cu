#include "hip/hip_runtime.h"
//#include <hip/hip_cooperative_groups.h>
#include "gpukv/fs_calls.cu.h"
#include "gpukv/host_loop.h"
#include "gpukv/util.cu.h"

#include "key-p2p.h"
#include "key_api.h"

#include "pr.cu.h"

// gardenia
#include "gardenia/timer.h"
#include "gardenia/common.h"

#define TOTAL_VERTICES 1971281
//#define TOTAL_VERTICES 20
//using namepspace cooperative_groups;

static __inline__ unsigned long long rdtsc(void){
    unsigned hi, lo;
    __asm__ __volatile__ ("rdtsc": "=a"(lo), "=d"(hi));
    return ((unsigned long long)lo | ( ((unsigned long long)hi) << 32) );
}

int main(int argc, char **argv){
	int num_thread = 1, handle = 1, call = 0;
	int device = 0;
	int gpu_thread = 1024, gpu_block = 28;
	int p2p_size = (192*1024*1024);
	int batch = 32;

	int tot_vert = TOTAL_VERTICES;
	unsigned int request_size=4096;
	int request_num, request_mode;

	while(1){
		int c;
		c=getopt(argc,argv,"s:n:m:p:i:b:t:c:q:h");
		if(c==-1) break;
		switch(c){
			case 'p':
				num_thread=strtol(optarg,NULL,0);
				break;
			case 'i':
				handle=strtol(optarg,NULL,0);
				break;
			case 'b':
				gpu_block=strtol(optarg,NULL,0);
				break;
			case 't':
				gpu_thread=strtol(optarg,NULL,0);
				break;
			case 'c':
				call=strtol(optarg,NULL,0);
				break;
			case 'q':
				batch=strtol(optarg,NULL,0);
				break;
			case 'v':
				tot_vert=strtol(optarg, NULL, 0);
				break;
			case 'h':
				printf("syntax: %s -p <num_polling_thread> -i <process:one-0,batch_async-1,batch_sync-2> -q <batch_size> -b <gpu block> -t < gpu theads in block> -c <call:0-thread,1-warp,2-block> -v <# of vertex> -h\n",argv[0]);
				exit(1);
				break;
			///// Unused
			case 's':
				request_size=strtol(optarg,NULL,0);
				break;
			case 'n':
				request_num=strtol(optarg,NULL,0);
				break;
			case 'm':
				request_mode=strtol(optarg,NULL,0);
				break;

			default:
				printf("ERROR: invalid option\n");
				exit(1);
				break;
		}
	}

	//////////// GPU setting

	CUDA_SAFE_CALL(hipSetDeviceFlags(hipDeviceMapHost));
	CUDA_SAFE_CALL(hipSetDevice(device));

	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
	PRINT_TIMES("GPU id: %d, name: %s\nPCI domain: %d, bus: %d, device: %d\n",device,deviceProp.name,deviceProp.pciDomainID,deviceProp.pciBusID,deviceProp.pciDeviceID);

	PRINT_TIMES("Running on device %d: \"%s\"\n", device, deviceProp.name);

	PRINT_TIMES("\tblocks %d threads %d\n", gpu_block, gpu_thread);
	////////////////////////////////////////////////
	////////////////////p2p memory mapping, and general memory
	void *dummy;
	ASSERTRT(hipMalloc(&dummy,0));

	hipDeviceptr_t gpu_memory;
	char* cuda_memory;
	char *cuda_put,*user_put;

	ASSERTDRV(hipMalloc(&gpu_memory, p2p_size));
	ASSERTDRV(hipMemsetD8(gpu_memory, 0x00, p2p_size));
	PRINT_TIMES("device ptr: 0x%lx \n",gpu_memory);
	unsigned int flag=1;
	ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, gpu_memory));
	//return 0;
	struct I_KEY_PIN_PARAM pin;
	struct I_KEY_UNPIN_PARAM unpin;
	int pin_fd;

	pin_fd=open(D,O_RDWR);
	pin.addr=gpu_memory;
	pin.size=p2p_size;
	pin.id=0;
	int ret=ioctl(pin_fd,I_KEY_PIN_BUF,&pin);
	if(ret!=0){printf("gpu pin error!\n"); return 0;}
	//printf("[DEBUG:gpukv_bfs] pinned!\n");
	PRINT_TIMES("info.va: %lx\n",pin.start_addr);   
	PRINT_TIMES("info.mapped_size: %lu B,(%lu MB)\n",pin.pin_size,pin.pin_size/(1024*1024));
	//      OUT << "info.page_size: " << info.page_size << endl;

	// remember that mappings start on a 64KB boundary, so let's
	// calculate the offset from the head of the mapping to the
	// beginning of the buffer
	int start_off = pin.start_addr-gpu_memory;//info.va - d_A;
	PRINT_TIMES("page offset: %lx\n", start_off);
	cuda_memory=(char*)gpu_memory;


	////////// KV open
	int key_ret=key_open(F_NAME);

	volatile GPUGlobals* gpuGlobals;
	initializer(&gpuGlobals,num_thread);
	gpuGlobals->user_put=user_put;
	gpuGlobals->cuda_put=cuda_put;

	int n=sqrt(gpu_thread);
	//dim3 nthreads(n,n);
	int nthreads = 256;

	int m = tot_vert;
	printf("[DEBUG:gpukv_bfs] Total vertices: %d\n", m);
	int *column_indices;
	int *degree;
	ScoreT *h_scores, *d_scores;
	ScoreT *d_contrib;
	ScoreT base_score = (1.0f - 0.85) / m;
	hipError_t error;

	/************ Kernel No finish version ***********/
	int max_req = 48*1024;
	int src = 0;
	int nblocks = 192;
	nthreads = BLOCK_SIZE;
	struct g_flag h_flag;
	h_flag.round = 0;
	h_flag.diff = 0.0f;
	struct g_flag *d_flag;
	float *d_diff;

	/////// Init distance
	h_scores = (ScoreT *)malloc(sizeof(ScoreT)*m);
	for(int i = 0; i < m; ++i)
		h_scores[i] = 1.0f/m;

	Timer t;
	t.Start();
    unsigned long long clock_start = rdtsc();
	/////// Data transmission & launch kernel
		CUDA_SAFE_CALL(hipMalloc((void**)&column_indices, nthreads*nblocks*128 * sizeof(int)));
		CUDA_SAFE_CALL(hipMalloc((void**)&degree, m * sizeof(int)));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_scores, m * sizeof(ScoreT)));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_contrib, m * sizeof(ScoreT)));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_flag, sizeof(struct g_flag)));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_diff, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpy(d_flag, &h_flag, sizeof(struct g_flag), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m*sizeof(ScoreT), hipMemcpyHostToDevice));

		/*pr_kernel<<<nblocks, nthreads>>>(cuda_memory, request_size, m, column_indices,
				degree, d_contrib, d_scores, base_score, d_flag);*/
		inner_kernel<<<1,1,0,gpuGlobals->streamMgr->kernelStream>>>(cuda_memory, request_size, m, max_req, column_indices,
				degree, d_contrib, d_scores, base_score, d_diff);


	///////// call gpufs handler	
	run_gpufs_handler(gpuGlobals, device,pin_fd, num_thread,handle,batch,call);
	
	error = hipDeviceSynchronize();

    unsigned long long clock_end = rdtsc();
	t.Stop();
	//end_complete_async(pin_fd);
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	error = hipDeviceSynchronize();
	/*for(int i = 0; i < (m > 100 ? 20 : m); i++){
		printf("dist[%d] = %u\n", i, h_dist[i]);
	}*/
	printf("Calculation time: %f ms. (%f secs.)\n", t.Millisecs(), t.Seconds());
    printf("clock: %llu\n", clock_end - clock_start);
	
	if (error != hipSuccess)
	{
		printf("Device failed, CUDA error message is: %s\n\n",
				hipGetErrorString(error));
	}

	PRINT_TIMES("\n");
	delete gpuGlobals;

	PRINT_TIMES("unpinning\n");
	ASSERT_EQ(ioctl(pin_fd,I_KEY_UNPIN_BUF,&unpin),0);
	ASSERTDRV(hipFree(gpu_memory));

	CUDA_SAFE_CALL(hipFree(column_indices));
	CUDA_SAFE_CALL(hipFree(degree));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_flag));
	free(h_scores);

	close(pin_fd);
	key_close(F_NAME);

	hipDeviceReset();

	return 0;
}

